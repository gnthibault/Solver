
// Local
#include "Field/CoeffContainer.h"

// STL
#include <vector>

// Local
#include "cuda/utils/ManagedAllocator.cu.h"


// Explicit instanciation
template class
CoeffContainer1D<float,std::vector<float,managedAllocator<float>>>;
template class
CoeffContainer2D<float,std::vector<float,managedAllocator<float>>>;
template class
CoeffContainer3D<float,std::vector<float,managedAllocator<float>>>;

template class
CoeffContainer1D<double,std::vector<double,managedAllocator<double>>>;
template class
CoeffContainer2D<double,std::vector<double,managedAllocator<double>>>;
template class
CoeffContainer3D<double,std::vector<double,managedAllocator<double>>>;

