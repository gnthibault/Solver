//Local
#include "cuda/utils/DeviceManager.cu.h"

//STL
#include <algorithm>
#include <cassert>
#include <numeric>
#include <iostream>
#include <iomanip>

//OMP
#include <omp.h>

//Local
#include <cuda/utils/utils.cu.h>

#define MIN_COMPUTE_CAPABILITY 3
#define NB_BYTE_PER_GIGA 1024*1024*1024

DeviceManager& DeviceManager::GetInstance() {
  static DeviceManager instance;
  return instance;
}

void DeviceManager::reset(size_t nbDevice) {
  //clear current device list
  m_vDeviceDesc.clear();

  //Use cuda API to count number of devices
  int maxNbDevice = 0;
  checkCudaErrors(hipGetDeviceCount(&maxNbDevice));

  //Build device list
  for (int devId=0; devId<maxNbDevice; devId++) {
    checkCudaErrors(hipSetDevice(devId));
    checkCudaErrors(hipDeviceReset());

    checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

    hipDeviceProp_t devProp;
    bool bComputeCompatibility=true;
    checkCudaErrors(hipGetDeviceProperties(&devProp,devId));

    if (devProp.major<=MIN_COMPUTE_CAPABILITY) {
      std::cout<<"Device "<<devId<<" compute capability is too old"<<std::endl;
      //TODO TN: hipDeviceGetByPCIBusId()
      bComputeCompatibility=false;
    }

    if( (m_vDeviceDesc.size()<nbDevice || nbDevice<=0) &&
        bComputeCompatibility) {
      checkCudaErrors(hipSetDevice(devId));
      //Build DeviceDesc
      DeviceDesc devDesc;
      devDesc.id=devId;
      devDesc.deviceProp=devProp;
      //By default, create only one stream
      hipStream_t cudaStream;
      checkCudaErrors(hipStreamCreate(&cudaStream));
      devDesc.vStream.push_back(cudaStream);
      //Finally push back new device descriptor in vector
      m_vDeviceDesc.push_back(devDesc);
    }
  }
}

const std::vector<DeviceDesc>& DeviceManager::GetDeviceDesc() const {
  return m_vDeviceDesc;
}

int DeviceManager::GetCurrentDeviceIdx() const {
  int devId;
  checkCudaErrors(hipGetDevice(&devId));

  auto it=std::find_if(m_vDeviceDesc.cbegin(), m_vDeviceDesc.cend(),
    [devId](const DeviceDesc& in) {return in.id==devId;} );

  assert(it!=m_vDeviceDesc.cend() );
  return std::distance(m_vDeviceDesc.cbegin(), it) ;
}

DeviceManager::~DeviceManager() {
  for (auto& it : m_vDeviceDesc ) {
    for (auto& stream : it.vStream) {
      checkCudaErrors(hipStreamDestroy(stream));
    }
  }
}

DeviceManager::DeviceManager() {
  reset();

  std::cout << "******************************"
    "**********************************"<<std::endl;
  std::cout << "*  ID                          "
    "Name                   Memory   *"<<std::endl;
  for (const auto& dev : m_vDeviceDesc) {
    std::cout<<"*  "<<dev.id<<std::setw(31)<<
      dev.deviceProp.name<<std::setw(25)<<
      dev.deviceProp.totalGlobalMem/NB_BYTE_PER_GIGA<<"Go *"<<std::endl;
  }
  std::cout << "*                              "
    "    *"<<std::endl;
  std::cout << "*******************************"
    "*********************************"<<std::endl;
}
